
#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE 4
#define BLOCK_SIZE 16

using namespace std;

typedef void (*dev_func_t)(int *element);

__device__ void mult_by_2(int *element) { //<=> callback
    *element *= 2;
}

__global__ void kernel(int *dev_matrix, dev_func_t dev_function) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x; //column
    if (idx < SIZE) {
        for (int i = 0; i < SIZE; ++i)
            (*dev_function)(&dev_matrix[SIZE * i + idx]);
    }
}

__host__ void error_handler(hipError_t err) {
    if (err != hipSuccess) {
        cerr << hipGetErrorString(err) << endl;
        exit(1);
    }   
}

__host__ void output(int (&matrix)[SIZE][SIZE]) {
    for (int i = 0; i < SIZE; ++i) {
        for (int j = 0; j < SIZE; ++j) {
            cout << matrix[i][j] << " ";
        }
        cout << endl;
    }
}

//static pointer to device function
__device__ dev_func_t p_func = mult_by_2;

__host__ void map() {

    int matrix[SIZE][SIZE] = {
        {1, 2, 3, 4},
        {5, 6, 7, 8},
        {9, 10, 11, 12},
        {13, 14, 15, 16}
    };

    cout << "Initial matrix" << endl;
    output(matrix);

    int *dev_matrix = NULL;

    dev_func_t h_func; //host pointer to device func

    error_handler(hipMalloc((void**)&dev_matrix, SIZE * SIZE * sizeof(int)));
    error_handler(hipMemcpy(dev_matrix, matrix, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice));
    error_handler(hipMemcpyFromSymbol(&h_func, HIP_SYMBOL(p_func), sizeof(dev_func_t)));

    dev_func_t callback = h_func;

    //kernel start
    //kernel<<<1, 1>>>(dev_matrix, myfunc); //test

    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE, (SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE);
    kernel<<<gridDim, blockDim>>>(dev_matrix, callback);

    error_handler(hipMemcpy(matrix, dev_matrix, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost));

    cout << endl << "After Map (multiply each element by 2):" << endl;
    output(matrix);

    error_handler(hipFree(dev_matrix));
}

__host__ int main() {

    map();
    return 0;
}