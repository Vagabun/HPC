#include <iostream>
#include <hip/hip_runtime.h>


#define N 10

void error_handler(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}

__global__ void kernel(int *a, int *b, int *c) {
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

__host__ int main() {
    int *a, *b, *c;
    int a_host[N], b_host[N], c_host[N];

    error_handler(hipMalloc((void**)&a, N * sizeof(int)));
    error_handler(hipMalloc((void**)&b, N * sizeof(int)));
    error_handler(hipMalloc((void**)&c, N * sizeof(int)));

    for (int i = 0; i < N; i++) {
        a_host[i] = i;
        b_host[i] = i * i;
    }

    error_handler(hipMemcpy(a, a_host, sizeof(int) * N, hipMemcpyHostToDevice));
    error_handler(hipMemcpy(b, b_host, sizeof(int) * N, hipMemcpyHostToDevice));

    kernel <<<N, 1>>> (a, b, c);
    hipDeviceSynchronize();

    error_handler(hipMemcpy(c_host, c, sizeof(int) * N, hipMemcpyDeviceToHost));

    std::cout << "first vector: ";
    for (int i = 0; i < N; i++) {
        std::cout << a_host[i] << " ";
    }
    std::cout << std::endl << "second vector: ";
    for (int i = 0; i < N; i++) {
        std::cout << b_host[i] << " ";
    }
    std::cout << std::endl << "result vector: ";
    for (int i = 0; i < N; i++) {
        std::cout << c_host[i] << " ";
    }
    std::cout << std::endl;

    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}
