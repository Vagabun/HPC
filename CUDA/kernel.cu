#include <iostream>
#include <hip/hip_runtime.h>


#define N 10


__global__ void kernel(int *a, int *b, int *c) {
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main() {
    int *a, *b, *c;
    int a_host[N], b_host[N], c_host[N];

    hipMalloc((void**)&a, N * sizeof(int));
    hipMalloc((void**)&b, N * sizeof(int));
    hipMalloc((void**)&c, N * sizeof(int));

    for (int i = 0; i < N; i++) {
        a_host[i] = i;
        b_host[i] = i * i;
    }

    hipMemcpy(a, a_host, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(b, b_host, sizeof(int) * N, hipMemcpyHostToDevice);

    kernel <<<N, 1>>> (a, b, c);
    hipDeviceSynchronize();

    hipMemcpy(c_host, c, sizeof(int) * N, hipMemcpyDeviceToHost);

    std::cout << "first vector: ";
    for (int i = 0; i < N; i++) {
        std::cout << a_host[i] << " ";
    }
    std::cout << std::endl << "second vector: ";
    for (int i = 0; i < N; i++) {
        std::cout << b_host[i] << " ";
    }
    std::cout << std::endl << "result vector: ";
    for (int i = 0; i < N; i++) {
        std::cout << c_host[i] << " ";
    }
    std::cout << std::endl;

    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}
