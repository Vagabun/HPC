
#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE 4
#define BLOCK_SIZE 16

using std::cout;
using std::endl;
using std::cerr;

__global__ void std_multiplication_kernel(int *dev_matrix_1, int *dev_matrix_2, int *dev_answer) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x; //column
    int idy = blockDim.y * blockIdx.y + threadIdx.y; //row

    if (idx < SIZE && idy < SIZE) {
        int sum = 0;
        for (int i = 0; i < SIZE; ++i) {
            sum += dev_matrix_1[idy * SIZE + i] * dev_matrix_2[SIZE * i + idx];
        }
        dev_answer[idy * SIZE + idx] = sum;
    }
}

__host__ void error_handler(hipError_t err) {
    if (err != hipSuccess) {
        cerr << hipGetErrorString(err) << endl;
        exit(1);
    }   
}

__host__ void output(int (&matrix)[SIZE][SIZE]) {
    for (int i = 0; i < SIZE; ++i) {
        for (int j = 0; j < SIZE; ++j) {
            cout << matrix[i][j] << " ";
        }
        cout << endl;
    }
}

__host__ void matrix_multiplication() {
    // const int matrix_1[SIZE][SIZE] = {
    //     {1, 0, 0, 0},
    //     {0, 1, 0, 0},
    //     {0, 0, 1, 0},
    //     {0, 0, 0, 1}
    // };

    const int matrix_1[SIZE][SIZE] = {
        {1, 2, 3, 4},
        {5, 6, 7, 8},
        {9, 10, 11, 12},
        {13, 14, 15, 16}
    };
    const int matrix_2[SIZE][SIZE] = {
        {1, 0, 0, 0},
        {0, 1, 0, 0},
        {0, 0, 1, 0},
        {0, 0, 0, 1}
    };
    int answer[SIZE][SIZE] = {};

    int *dev_matrix_1 = NULL;
    int *dev_matrix_2 = NULL;
    int *dev_answer = NULL;

    //allocation without pitching
    error_handler(hipMalloc((void**)&dev_matrix_1, SIZE * SIZE * sizeof(int)));
    error_handler(hipMalloc((void**)&dev_matrix_2, SIZE * SIZE * sizeof(int)));
    error_handler(hipMalloc((void**)&dev_answer, SIZE * SIZE * sizeof(int)));

    //copy data from host to device
    error_handler(hipMemcpy(dev_matrix_1, matrix_1, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice));
    error_handler(hipMemcpy(dev_matrix_2, matrix_2, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice));

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE, (SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE);
    std_multiplication_kernel<<<gridDim, blockDim>>>(dev_matrix_1, dev_matrix_2, dev_answer);

    error_handler(hipMemcpy(answer, dev_answer, SIZE * SIZE * sizeof(int), hipMemcpyDeviceToHost));

    output(answer);

    error_handler(hipFree(dev_matrix_1));
    error_handler(hipFree(dev_matrix_2));
    error_handler(hipFree(dev_answer));
}

__host__ int main() {

    matrix_multiplication();

    return 0;
}