
#include <hip/hip_runtime.h>
#include <iostream>

// #define SIZE 128 //matrix height/width, for std_kernel
#define SIZE 4 //matrix height/width, for shared_kernel
#define BLOCK_SIZE 2 //threads per block, for shared_kernel

using std::cout;
using std::endl;
using std::cerr;

__global__ void std_kernel(int *matrix, int *vector, int *answer) {
    int t_id = blockDim.x * blockIdx.x + threadIdx.x; // t_id (thread id) <=> column id
    int sum = 0;
    if (t_id < SIZE) {
        for (int i = 0; i < SIZE; ++i)
            sum += matrix[SIZE * i + t_id] * vector[i];
        answer[t_id] = sum;
    }
}

__host__ void error_handler(hipError_t err) {
    if (err != hipSuccess) {
        cerr << hipGetErrorString(err) << endl;
        exit(1);
    }   
}

//init identity matrix
__host__ void init_matrix(int (&matrix)[SIZE][SIZE]) {
    for (int i = 0; i < SIZE; ++i)
        for (int j = 0; j < SIZE; ++j)
           if (i == j)
               matrix[i][j] = 1;
           else
               matrix[i][j] = 0;
}

__host__ void init_vector(int (&vector)[SIZE]) {
    int n = 1;
    for (int i = 0; i < SIZE; ++i)
        vector[i] = n++;
}

//transpose and transform matrix into vector
__host__ void transpose_matrix(int (&matrix)[SIZE][SIZE], int (&linear_matrix)[SIZE * SIZE]) {
    for (int i = 0; i < SIZE; ++i)
        for (int j = 0; j < SIZE; ++j)
            linear_matrix[SIZE * j + i] = matrix[i][j];
}

__host__ void output(int (&answer)[SIZE]) {
    for (int i = 0; i < SIZE; ++i)
        cout << answer[i] << " ";
    cout << endl;
}

__host__ void std_matrix_vector_multiplication() {
    int matrix[SIZE][SIZE] = { {} };
    int vector[SIZE] = {};
    int linear_matrix[SIZE * SIZE] = {};
    int answer[SIZE] = {};

    init_matrix(matrix);
    init_vector(vector);
    transpose_matrix(matrix, linear_matrix);

    int *dev_linear_matrix = NULL;
    int *dev_vector = NULL;
    int *dev_answer = NULL;

    error_handler(hipMalloc((void**)&dev_linear_matrix, SIZE * SIZE * sizeof(int)));
    error_handler(hipMalloc((void**)&dev_vector, SIZE * sizeof(int)));
    error_handler(hipMalloc((void**)&dev_answer, SIZE * sizeof(int)));

    error_handler(hipMemcpy(dev_linear_matrix, linear_matrix, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice));
    error_handler(hipMemcpy(dev_vector, vector, SIZE * sizeof(int), hipMemcpyHostToDevice));

    //const int num_blocks = 1;
    //dim3 threads_per_block(32);
    //std_kernel<<<num_blocks, threads_per_block>>>(dev_linear_matrix, dev_vector, dev_answer);
    
    const int threads_per_block = 32;
    std_kernel<<<(SIZE + (threads_per_block - 1))/threads_per_block, threads_per_block>>>(dev_linear_matrix, dev_vector, dev_answer);

    error_handler(hipMemcpy(answer, dev_answer, SIZE * sizeof(int), hipMemcpyDeviceToHost));
    output(answer);

    error_handler(hipFree(dev_linear_matrix));
    error_handler(hipFree(dev_vector));
    error_handler(hipFree(dev_answer));
}

__global__ void shared_kernel(int *dev_matrix, int *dev_vector, int *dev_answer) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int sum = 0;
    __shared__ int vector_cache[SIZE];

    for (int i = 0; i < (SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE; ++i) { //(SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE - total size of blocks 
        int stride = BLOCK_SIZE * i; //offset 
        vector_cache[threadIdx.x] = dev_vector[stride + threadIdx.x];
        __syncthreads();
        for (int j = 0; j < BLOCK_SIZE; ++j) {
            if (stride + j < SIZE)
                sum += dev_matrix[(stride + j) * SIZE + idx] * vector_cache[j];
        }
        __syncthreads();
    }
    dev_answer[idx] = sum;
}

__host__ void shared_matrix_vector_multiplication() {
    // const int matrix[SIZE][SIZE] = {
    //     {1, 0, 0, 0},
    //     {0, 1, 0, 0},
    //     {0, 0, 1, 0},
    //     {0, 0, 0, 1}
    // };
    const int matrix[SIZE][SIZE] = {
        {1, 5, 9, 13},
        {2, 6, 10, 14},
        {3, 7, 11, 15},
        {4, 8, 12, 16}
    };
    //const int vector[SIZE] = {1, 2, 3, 4};
    const int vector[SIZE] = {1, 1, 1, 1};
    int answer[SIZE] = {};

    int *dev_matrix = NULL;
    int *dev_vector = NULL;
    int *dev_answer = NULL;

    //allocate memory on device
    error_handler(hipMalloc((void**)&dev_matrix, SIZE * SIZE * sizeof(int)));
    error_handler(hipMalloc((void**)&dev_vector, SIZE * sizeof(int)));
    error_handler(hipMalloc((void**)&dev_answer, SIZE * sizeof(int)));

    //copy data from host to device
    error_handler(hipMemcpy(dev_matrix, matrix, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice));
    error_handler(hipMemcpy(dev_vector, vector, SIZE * sizeof(int), hipMemcpyHostToDevice));

    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((SIZE + blockDim.x - 1)/blockDim.x);
    shared_kernel<<<gridDim, blockDim>>>(dev_matrix, dev_vector, dev_answer);
    
    //const int threads_per_block = 2;
    //shared_kernel<<<(SIZE + (threads_per_block - 1))/threads_per_block, threads_per_block>>>(dev_linear_matrix, dev_vector, dev_answer);

    error_handler(hipMemcpy(answer, dev_answer, SIZE * sizeof(int), hipMemcpyDeviceToHost));
    output(answer);

    error_handler(hipFree(dev_matrix));
    error_handler(hipFree(dev_vector));
    error_handler(hipFree(dev_answer));
}

__host__ int main() {

    //std_matrix_vector_multiplication();
    shared_matrix_vector_multiplication();

    return 0;
}
