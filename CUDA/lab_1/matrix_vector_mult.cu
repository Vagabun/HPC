
#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE 128 //matrix height * width

using std::cout;
using std::endl;
using std::cerr;

__global__ void kernel(int *matrix, int *vector, int *answer) {
    int t_id = blockDim.x * blockIdx.x + threadIdx.x; // t_id (thread id) <=> column id
    int sum = 0;
    if (t_id < SIZE) {
        for (int i = 0; i < SIZE; ++i)
            sum += matrix[SIZE * i + t_id] * vector[i];
        answer[t_id] = sum;
    }
}

__host__ void error_handler(hipError_t err) {
    if (err != hipSuccess) {
        cerr << hipGetErrorString(err) << endl;
        exit(1);
    }   
}

//init identity matrix
__host__ void init_matrix(int (&matrix)[SIZE][SIZE]) {
    for (int i = 0; i < SIZE; ++i)
        for (int j = 0; j < SIZE; ++j)
           if (i == j)
               matrix[i][j] = 1;
           else
               matrix[i][j] = 0;
}

__host__ void init_vector(int (&vector)[SIZE]) {
    int n = 1;
    for (int i = 0; i < SIZE; ++i)
        vector[i] = n++;
}

//transpose and transform matrix into vector
__host__ void transpose_matrix(int (&matrix)[SIZE][SIZE], int (&linear_matrix)[SIZE * SIZE]) {
    for (int i = 0; i < SIZE; ++i)
        for (int j = 0; j < SIZE; ++j)
            linear_matrix[SIZE * j + i] = matrix[i][j];
}

__host__ void output(int (&answer)[SIZE]) {
    for (int i = 0; i < SIZE; ++i)
        cout << answer[i] << " ";
    cout << endl;
}

__host__ void matrix_vector_multiplication() {
    int matrix[SIZE][SIZE] = { {} };
    int vector[SIZE] = {};
    int linear_matrix[SIZE * SIZE] = {};
    int answer[SIZE] = {};

    init_matrix(matrix);
    init_vector(vector);
    transpose_matrix(matrix, linear_matrix);

    int *dev_linear_matrix = NULL;
    int *dev_vector = NULL;
    int *dev_answer = NULL;

    error_handler(hipMalloc((void**)&dev_linear_matrix, SIZE * SIZE * sizeof(int)));
    error_handler(hipMalloc((void**)&dev_vector, SIZE * sizeof(int)));
    error_handler(hipMalloc((void**)&dev_answer, SIZE * sizeof(int)));

    error_handler(hipMemcpy(dev_linear_matrix, linear_matrix, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice));
    error_handler(hipMemcpy(dev_vector, vector, SIZE * sizeof(int), hipMemcpyHostToDevice));

    //const int num_blocks = 1;
    //dim3 threads_per_block(32);
    //kernel<<<num_blocks, threads_per_block>>>(dev_linear_matrix, dev_vector, dev_answer);
    const int threads_per_block = 32;
    kernel<<<(SIZE + (threads_per_block - 1))/threads_per_block, threads_per_block>>>(dev_linear_matrix, dev_vector, dev_answer);

    error_handler(hipDeviceSynchronize());
    error_handler(hipMemcpy(answer, dev_answer, SIZE * sizeof(int), hipMemcpyDeviceToHost));
    output(answer);

    error_handler(hipFree(dev_linear_matrix));
    error_handler(hipFree(dev_vector));
    error_handler(hipFree(dev_answer));
}

__host__ int main() {

    matrix_vector_multiplication();

    return 0;
}
