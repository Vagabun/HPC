
#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE 3 //matrix height * width

using std::cout;
using std::endl;
using std::cerr;

__global__ void kernel(int *matrix, int *vector, int *answer) {
    int column = threadIdx.x;
    int sum = 0;
    for (int i = 0; i < SIZE; ++i) {
        sum += matrix[SIZE * i + column] * vector[i];
    }
    answer[column] = sum;
}

__host__ void error_handler(hipError_t err) {
    if (err != hipSuccess) {
        cerr << hipGetErrorString(err) << endl;
        exit(1);
    }   
}

__host__ void init_matrix(int (&matrix)[SIZE][SIZE]) {
    //int n = 1;
    for (int i = 0; i < SIZE; ++i) {
        for (int j = 0; j < SIZE; ++j) {
            //matrix[i][j] = n++;
           if (i == j)
               matrix[i][j] = 1;
           else
               matrix[i][j] = 0;
        }
    }
}

__host__ void init_vector(int (&vector)[SIZE]) {
    int n = 1;
    for (int i = 0; i < SIZE; ++i)
        vector[i] = n++;
}

//transpose and transform matrix into vector
__host__ void transpose_matrix(int (&matrix)[SIZE][SIZE], int (&flat_matrix)[SIZE * SIZE]) {
    for (int i = 0; i < SIZE; ++i) {
        for (int j = 0; j < SIZE; ++j) {
            flat_matrix[SIZE * j + i] = matrix[i][j];
        }
    }
}

__host__ void matrix_vector_multiplication() {
    int matrix[SIZE][SIZE] = { {} };
    int vector[SIZE] = {};
    int flat_matrix[SIZE * SIZE] = {};
    int answer[SIZE] = {};

    init_matrix(matrix);
    init_vector(vector);
    transpose_matrix(matrix, flat_matrix);

    int *dev_flat_matrix = NULL;
    int *dev_vector = NULL;
    int *dev_answer = NULL;

    error_handler(hipMalloc((void**)&dev_flat_matrix, SIZE * SIZE * sizeof(int)));
    error_handler(hipMalloc((void**)&dev_vector, SIZE * sizeof(int)));
    error_handler(hipMalloc((void**)&dev_answer, SIZE * sizeof(int)));

    error_handler(hipMemcpy(dev_flat_matrix, flat_matrix, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice));
    error_handler(hipMemcpy(dev_vector, vector, SIZE * sizeof(int), hipMemcpyHostToDevice));

    int num_blocks = 1;
    dim3 threads_per_block(3);
    kernel<<<num_blocks, threads_per_block>>>(dev_flat_matrix, dev_vector, dev_answer);

    error_handler(hipDeviceSynchronize());

    error_handler(hipMemcpy(answer, dev_answer, SIZE * sizeof(int), hipMemcpyDeviceToHost));

    for (int i = 0; i < SIZE; ++i)
        cout << answer[i] << " ";
    cout << endl;

}

__host__ int main() {

    matrix_vector_multiplication();

    return 0;
}
